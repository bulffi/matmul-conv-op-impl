#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <chrono>
#include "gpu_op.h"

__global__ void single_element(const double* M, const double* N, double* out, unsigned long size, unsigned long n_y) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        sum += M[threadIdx.x * size + i] * N[i * n_y + threadIdx.y];
    }
    out[threadIdx.x * n_y + threadIdx.y] = sum;
}

__global__ void block_element(const double* M, const double* N, double* out, unsigned long m_x, unsigned long m_y,
                              unsigned long n_y, unsigned long x_range, unsigned long y_range) {
    int x_start = threadIdx.x * x_range;
    int y_start = threadIdx.y * y_range;
    int x_end = 0;
    int y_end = 0;
    if (threadIdx.x == blockDim.x - 1) {
        x_end = m_x;
    } else {
        x_end = x_start + x_range;
    }
    if (threadIdx.y == blockDim.y - 1) {
        y_end = n_y;
    } else {
        y_end = y_start + y_range;
    }
    for (int i = x_start; i < x_end; i++) {
        for (int j = y_start; j < y_end; j++) {
            double tempt_sum = 0;
            for (int k = 0; k < m_y; k++) {
                tempt_sum += M[i * m_y + k] * N[k * n_y + j];
            }
            out[i * n_y + j] = tempt_sum;
        }
    }
}

pybind11::array_t<double> gpu_matmul_base(pybind11::array_t<double> M, pybind11::array_t<double> N) {
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    auto m = M.unchecked<2>();
    auto n = N.unchecked<2>();
    std::size_t m_x = m.shape(0);
    std::size_t m_y = m.shape(1);
    std::size_t n_x = n.shape(0);
    std::size_t n_y = n.shape(1);
    std::size_t m_size = m_x * m_y;
    std::size_t n_size = n_x * n_y;
    std::size_t output_size = m_x * n_y;
    auto *result = new double[output_size];

    /// =========================================
    // in this initial implementation, we try to use as many threads as we can!
    // in Turing architecture, we can have 1024(32*32) threads per block
    // so we divide it like this.
    double* d_M;
    double* d_N;
    double* d_out;
    hipMalloc(&d_M, sizeof(double) * m_size);
    hipMalloc(&d_N, sizeof(double) * n_size);
    hipMalloc(&d_out, sizeof(double) * output_size);
    hipMemcpy(d_M, m.data(0,0), sizeof(double) * m_size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, n.data(0,0), sizeof(double) * n_size, hipMemcpyHostToDevice);
    std::chrono::steady_clock::time_point real_begin = std::chrono::steady_clock::now();
    if (m_x <= 32 && n_y <= 32) {
        dim3 threadDim{static_cast<unsigned int>(m_x), static_cast<unsigned int>(n_y), 1};
        single_element<<<1, threadDim>>>(d_M, d_N, d_out, m_y, n_y);
    } else {
        std::size_t x_range = m_x / 32;
        std::size_t y_range = n_y / 32;
        dim3 threadDim{
            static_cast<unsigned int>(std::min((unsigned long)32, m_x)),
            static_cast<unsigned int>(std::min((unsigned long)32, n_y)),
            1
        };
        block_element<<<1, threadDim>>>(d_M, d_N, d_out, m_x, m_y, n_y, x_range, y_range);
    }
    hipDeviceSynchronize();
    hipMemcpy(result, d_out, sizeof(double) * output_size, hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_N);
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time " <<  std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << std::endl;
    std::cout << "Real Time " <<  std::chrono::duration_cast<std::chrono::milliseconds>(end - real_begin).count() << std::endl;
    /// =========================================

    pybind11::capsule free_when_done(result, [](void *f) {
        auto *foo = reinterpret_cast<double *>(f);
        delete[] foo;
    });
    return pybind11::array_t<double> {
            {m_x, n_y},
            result,
            free_when_done
    };
}

pybind11::array_t<double> gpu_matmul_multi_sm(pybind11::array_t<double> M, pybind11::array_t<double> N) {
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    auto m = M.unchecked<2>();
    auto n = N.unchecked<2>();
    std::size_t m_x = m.shape(0);
    std::size_t m_y = m.shape(1);
    std::size_t n_x = n.shape(0);
    std::size_t n_y = n.shape(1);
    std::size_t m_size = m_x * m_y;
    std::size_t n_size = n_x * n_y;
    std::size_t output_size = m_x * n_y;
    auto *result = new double[output_size];

    /// =========================================
    // in this initial implementation, we try to use as many threads as we can!
    // in Turing architecture, we can have 1024(32*32) threads per block
    // so we divide it like this.
    double* d_M;
    double* d_N;
    double* d_out;
    hipMalloc(&d_M, sizeof(double) * m_size);
    hipMalloc(&d_N, sizeof(double) * n_size);
    hipMalloc(&d_out, sizeof(double) * output_size);
    hipMemcpy(d_M, m.data(0,0), sizeof(double) * m_size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, n.data(0,0), sizeof(double) * n_size, hipMemcpyHostToDevice);
    std::chrono::steady_clock::time_point real_begin = std::chrono::steady_clock::now();



    hipDeviceSynchronize();
    hipMemcpy(result, d_out, sizeof(double) * output_size, hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_N);
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time " <<  std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << std::endl;
    std::cout << "Real Time " <<  std::chrono::duration_cast<std::chrono::milliseconds>(end - real_begin).count() << std::endl;
    /// =========================================

    pybind11::capsule free_when_done(result, [](void *f) {
        auto *foo = reinterpret_cast<double *>(f);
        delete[] foo;
    });
    return pybind11::array_t<double> {
            {m_x, n_y},
            result,
            free_when_done
    };
}


PYBIND11_MODULE(gpu_op, m){
    m.doc() = "matmul & conv with gpu";
    m.def("gpu_matmul_base", &gpu_matmul_base, "multiply 2 matrix");
    m.def("gpu_matmul_multi_sm", &gpu_matmul_multi_sm, "multiply 2 matrix using many SMs");
}